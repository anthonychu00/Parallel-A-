#include "hip/hip_runtime.h"
#include <assert.h>
#include <hiprand.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
__device__ __host__ void shiftqueue(int* array_sh,int dim,int k);
__device__ __host__ int checkIfQueueEmpty(int* array_sh, int dim, int k);
__device__ __host__ int heuristic(int dim, int current, int end);
__device__ int findNextInsertionPoint(int * sizes, int k);

__global__ void warmup(){
printf("bloop\n");
}




//store a size array somewhere
__global__ void traverse(int * grid, int start, int end, int dim, int k){//k is number of queues
									//6 and 5 in test example
	__shared__ int prevNode [36];//previous Node to print out route
	__shared__ int lowestCost [36];//current lowest cost to get to that Node
	__shared__ int array [30];//simulates priority queues, k marker at each step (dim*k)=(16*5)
	__shared__ int heuristics[30]; //will store the heuristics corresponding to a
	__shared__ int sizes [5];//stores current sizes of priority queues
	__shared__ int expandedNodes[20];//k*4 for 4 directions
	int extracted;
	int flag=0;
	lowestCost[end] =0;
	
	
		
	//presets shared memory
	if (threadIdx.x == 0)
	{
		for(int i = 0; i<dim*k;i++)
		{
			array[i] = -1;	
		}
		
		for(int i = 0; i<dim*dim;i++)
		{
			lowestCost[i] = 2000;//whatever max int is
			
		}
		for(int i = 0; i<k;i++)
		{
			sizes[i] = 0;
		}
		for(int i =0; i<4*k;i++)
		{
			expandedNodes[i]=-1;
		}
	}

	__syncthreads();
	
		

	array[0]=start;
	sizes[0]=1;
	lowestCost[start] = 0;

	
	while(checkIfQueueEmpty(array,dim,k)!=0)
	{
		
		if(flag==1)
		{
			break;
		}
		//stuff expandedNodes array
		if(array[dim*threadIdx.x]!=-1)//check if corresponding priority queue is empty
		{
			extracted=array[dim*threadIdx.x];//set the extracted Node to the front 										of this P.Queue
			shiftqueue(array,dim,threadIdx.x);
			sizes[threadIdx.x]--;
			//extraction and other stuff here
			

			if(extracted == end||flag ==1)
			{
				flag = 1;
				break;
			}
			
			
			//add to expanded nodes here
			int top = extracted-dim;
			int bottom = extracted+dim;
			int left = extracted-1;
			int right = extracted+1;
			if(extracted%dim==0)
				left = -1;
			if(extracted%dim==dim-1)
				right = -1;
			
			
			//dumps adjacent squares into array
			expandedNodes[threadIdx.x*4]=top;
			expandedNodes[threadIdx.x*4+1]=bottom;
			expandedNodes[threadIdx.x*4+2]=left;
			expandedNodes[threadIdx.x*4+3]=right;

			for(int i =0; i<4;i++)
			{
				printf("%d ",expandedNodes[threadIdx.x*4+i]);
			}
				printf("Before dedup \n ");
			//__syncthreads();
			
			//checks adjacent squares
			//deduplicates list
			for(int i =0;i<4;i++)
			{
				int curNum = expandedNodes[threadIdx.x*4+i];
				
			      if(curNum<0 || curNum>dim*dim|| grid[curNum]==0||lowestCost[extracted]+1>lowestCost[curNum])
				{					
					expandedNodes[threadIdx.x*4+i]=-1;
					continue;
				}//checks for invalid indices
			

				if(expandedNodes[threadIdx.x*4+i]!=-1)
				{
					
					//route is shorter, therefore update cost and previous node
					lowestCost[curNum]=lowestCost[extracted]+1;
					prevNode[curNum]=extracted;
				}
				

				
			}
			for(int i =0; i<4;i++)
			{
				printf("%d ",expandedNodes[threadIdx.x*4+i]);
			}	
				printf("After dedup \n ");
			
			//__syncthreads();

			//start heuristic of Nodes not -1 in expandedNodes 
			
			/*for(int i=0;i<4;i++)
			{
				
				
				if(expandedNodes[i]!=-1)
				{
					int h = lowestCost[expandedNodes[i]]+				 							heuristic(dim,expandedNodes[i],end);
					int check =0; 
//int targetLocation=findNextInsertionPoint(sizes, k);
//int l = atomicCAS(&array[targetLocation*dim+sizes[targetLocation]], -1,expandedNodes[i]);
//printf("T: %d\n", targetLocation);
//printf("Sizes: %d\n", sizes[targetLocation]);
					while(check==0)
					{
						int targetLocation=findNextInsertionPoint(sizes, k);
						printf("T: %d\n", targetLocation);
						printf("Sizes: %d\n", sizes[targetLocation]);
				printf("Loc = %d\n", array[targetLocation*dim+sizes[targetLocation]]);
						if(atomicCAS(&array[targetLocation*dim+
							sizes[targetLocation]], 							-1,expandedNodes[i])==-1)
						{
							printf("Got here \n");
				  		heuristics[targetLocation*dim+sizes[targetLocation]]= h;
							check = 1;
							sizes[targetLocation]++;
						}//check ordering later
					}
				
				}
								
			}*/

			if(threadIdx.x==0)
			{
				for(int i=0;i<4*k;i++)
				{
					if(expandedNodes[i]==-1)
						continue;
					int targetLocation=findNextInsertionPoint(sizes, k);
							array[targetLocation*dim+sizes[targetLocation]]=expandedNodes[i];
					sizes[targetLocation]++;
					
				}
			}

			
			__syncthreads();
				
		}//end of the larger if statement, coded this way to prevent warp divergence
		
		__syncthreads();

	}//end of while loop, shoould have found route or not by here
	if(threadIdx.x==0)
	printf("prev : \n%d\n",prevNode[end]);
}

__device__ int findNextInsertionPoint(int * sizes, int k){
	
	int smallestQueue = -100;
	int curSmallest = 5000;
	for(int i = 0; i<k;i++)
	{
		if(sizes[i]<curSmallest)
		{
			smallestQueue = i;
			curSmallest = sizes[i];
		}
	} 
	
	return smallestQueue;

}

__device__ __host__ int heuristic(int dim, int current, int end){//heurstics (Manhattan distance)
	
	int curX = current % dim;
	int curY = current / dim;
	int endX = end % dim;
	int endY = end / dim;
	
	return (int)(fabsf(curX-endX) + fabsf(curY-endY));
	
}

 __device__  void shiftqueue(int* array_sh,int dim,int k)
{
	
	for(int i=0;i<dim;i++)
	{
		if(i==dim-1)
			array_sh[dim*k+i] = -1;
		else
			array_sh[dim*k+i]= array_sh[dim*k+i+1];
		
	}
}

__device__  int checkIfQueueEmpty(int* array_sh, int dim, int k)
{
	int check = 0;//0 if all queues empty, 1 otherwise
	for(int i=0;i<k;i++)
	{
		if(array_sh[dim*i] != -1){
			check = 1;
			break;
		}
	}
	return check;
}

__device__ __host__ void print_board(int *game, int dim){

	for (int y = 0; y < dim; y++){
                for (int x = 0; x < dim; x++){
			printf("%d ", game[x + dim*y]);
		}
		printf("\n");
	}
}

__global__ void test()
{
	int i = threadIdx.x;
	printf("%d\n",i);
}
int main () {
	time_t t;
  	//int num_iter = 10;
	int dim = 6;
	int *grid = (int *) malloc(dim*dim*sizeof(int));
        srand((unsigned) time(&t));
	double p = 0.5;

	for (int i = 0; i < dim*dim; i++){
		grid[i] = p < (double)rand()/(double)(RAND_MAX);
		//old_game[i]=1;
	}
	int * dev_grid;
	hipMalloc((void**)&dev_grid, dim*dim*sizeof(int));
	hipMemcpy(dev_grid,grid,dim*dim*sizeof(int),hipMemcpyHostToDevice);

	print_board(grid, dim);	
	//1s are spaces you can walk on
	int startPoint = -1;
	while (startPoint == - 1){
		int rando = rand()%(dim*dim);//dim*dim - 1 is highest number( 0 to dim*dim-1)
		if (grid[rando] ==1)
			startPoint = rando;
	}
	
	int endPoint = -1;
	while (endPoint == - 1){
		int rando = rand()%(dim*dim);
		if (grid[rando] ==1&&rando!=startPoint)
			endPoint = rando;
	}
	
	printf("%d %d\n",startPoint,endPoint);

	traverse<<<1,5>>>(dev_grid,startPoint,endPoint,dim,5);//last value is # of p-queues
	//printf("%d \n", heuristic(6,18,10));
	//test<<<1,20>>>();
	hipDeviceSynchronize();
	return 0;


}
