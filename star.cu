#include "hip/hip_runtime.h"
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//You need to compile the program as "nvcc -G star.cu"
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
#include <assert.h>
#include <hiprand.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "timerc.h"
__device__ __host__ void shiftqueue(int* array_sh, int dim,int k);
__device__ __host__ int checkIfQueueEmpty(int * array_sh, int dim, int k);
__device__ __host__ int heuristic(int dim, int current, int end);
__device__ int findNextInsertionPoint(int * sizes, int k);
__device__ int duplicateAdjacents(int * adjacents, int a, int k, int currentPos);
__device__ void organizeQueue(int * queue, int targetLocation, int * heuristics, int h, int which, int dim);
__device__ __host__ void print_board(int * game, int dim);

__global__ void traverse(int * grid, int start, int end, int dim, int k, int * result){//k is number of queues
									//6 and 5 in this example
	__shared__ int prevNode [36];//previous Node to print out route
	__shared__ int lowestCost [36];//current lowest cost to get to that Node
	__shared__ int array [30];//simulates priority queues, k marker at each step (dim*k)=(16*5)
	__shared__ int heuristics[30]; //will store the heuristics corresponding to a
	__shared__ int sizes [5];//stores current sizes of priority queues
	__shared__ int flag;
	__shared__ int expandedNodes[20];//k*4 for 4 directions
	int extracted;
	flag = 0;
	lowestCost[end] = 0;
	prevNode[end] = -5;
	
		
	//presets shared memory
	if (threadIdx.x == 0)
	{
		for(int i = 0; i < dim * k ;i++)
		{
			array[i] = -1;	
		}
		
		for(int i = 0; i < dim * dim ;i++)
		{
			lowestCost[i] = 2000;//whatever max int is
			
		}
		for(int i = 0; i < k ; i++)
		{
			sizes[i] = 0;
		}
		for(int i =0; i < 4 * k;i++)
		{
			expandedNodes[i] = -1;
		}
	}

	__syncthreads();
	
		

	array[0] = start;
	sizes[0] = 1;
	lowestCost[start] = 0;

	
	while(checkIfQueueEmpty(array, dim, k) != 0)
	{	

		if(flag == 1)
		{
			break;
		}

		__syncthreads();
		
		
		//stuff expandedNodes array
		if(array[dim * threadIdx.x] != -1)//check if corresponding priority queue is empty
		{
			extracted = array[dim * threadIdx.x];//set the extracted Node to the front 										of this P.Queue
			printf("Extracted %d in %d\n", extracted, threadIdx.x);
			shiftqueue(array, dim, threadIdx.x);
			sizes[threadIdx.x]--;
			//extraction and other stuff here
			

			if(extracted == end || flag == 1)
			{
				flag = 1;
				break;
			}
			
			
			//add to expanded nodes here
			int top = extracted-dim;
			int bottom = extracted+dim;
			int left = extracted-1;
			int right = extracted+1;
			if(extracted % dim == 0)
				left = -1;
			if(extracted % dim == dim-1)
				right = -1;
			
			
			//dumps adjacent squares into array
			expandedNodes[threadIdx.x * 4 + 0] = top;
			expandedNodes[threadIdx.x* 4 + 1] = bottom;
			expandedNodes[threadIdx.x* 4 + 2] = left;
			expandedNodes[threadIdx.x* 4 + 3] = right;		
			
			//checks adjacent squares
			//deduplicates list
			for(int i = 0; i < 4 ;i++)
			{
				int curNum = expandedNodes[threadIdx.x * 4 + i];
				
			      if(curNum<0 || curNum>dim*dim|| grid[curNum]==0||lowestCost[extracted]+1>lowestCost[curNum])
				{					
					expandedNodes[threadIdx.x * 4 + i] =- 1;
					continue;
				}//checks for invalid indices
			

				if(expandedNodes[threadIdx.x * 4 + i] != -1)
				{
					
					//route is shorter, therefore update cost and previous node
					lowestCost[curNum] = lowestCost[extracted] + 1;
					prevNode[curNum] = extracted;
				}
				

				
			}
			//printf("After dedup: %d %d %d %d in thread %d\n ",expandedNodes[threadIdx.x*4+0],expandedNodes[threadIdx.x*4+1],expandedNodes[threadIdx.x*4+2],expandedNodes[threadIdx.x*4+3],threadIdx.x);
			
			

			//start heuristic of Nodes not -1 in expandedNodes 
			
			for(int i = 0;i < 4 ; i++)
			{
				int r = duplicateAdjacents(expandedNodes, expandedNodes[threadIdx.x * 4 + i],k,threadIdx.x * 4 + i);
				if(r != -1)
				{
					atomicExch(&expandedNodes[r], -1);
				}
				
				if(expandedNodes[threadIdx.x*4+i] != -1)
				{
					
					int h = lowestCost[expandedNodes[threadIdx.x*4+i]]+				 							heuristic(dim,expandedNodes[threadIdx.x*4+i],end);
					int check = 0; 
					while(check == 0)
					{
						int targetLocation = findNextInsertionPoint(sizes, k);
						if(atomicCAS(&array[targetLocation * dim + sizes[targetLocation]], -1, expandedNodes[threadIdx.x * 4 + i]) == -1)
						{
							
							heuristics[targetLocation * dim + sizes[targetLocation]] = h;
							sizes[targetLocation]++;	
							organizeQueue(array, targetLocation, heuristics, h, sizes[targetLocation], dim);
							check = 1;
							
						}
					}
				
				}
								
			}			
			
				
		}//end of the larger if statement, coded this way to prevent warp divergence
		
		__syncthreads();

	}//end of while loop, shoould have found route or not by here
	if(threadIdx.x == 0){
		printf("prev : \n%d\n", prevNode[end]);
		int current = end;	
		while(current != start)
		{
			if(prevNode[current] == -5)
				break;
			grid[current] = -1;
			current = prevNode[current];
		}
		if(current == start)
		{
			grid[current] = -1;
			printf("The path is marked by -1s\n");
			print_board(grid, dim);
		}
		else
			printf("No route found!\n");
	
	}

}

__device__ void organizeQueue(int * queue, int targetLocation, int * heuristics, int h, int which, int dim)
{
	int temp1;
	int temp2;
	for(int i = which-1; i >= 0 ;i--)
	{	
		if(i < 0)
			break;
		 if(heuristics[dim * targetLocation + i] > h)
		{
			temp1 = heuristics[dim * targetLocation + i];
			temp2 = queue[dim * targetLocation + i];
			heuristics[dim * targetLocation + i] = h;
			queue[dim * targetLocation + i] = queue[dim * targetLocation + which];
			queue[dim * targetLocation + which] = temp2;
			heuristics[dim * targetLocation + which] = temp1;
			which--;
		}

	}
}

__device__ int duplicateAdjacents(int * adjacents, int a, int k, int currentPos)
{
	for(int i = 1; i < 4 * k ;i++)
	{
		if(adjacents[i] == a && i != currentPos)
		{
			return i;
		}
	}
	return -1;
}

__device__ int findNextInsertionPoint(int * sizes, int k){
	
	int smallestQueue = -100;
	int curSmallest = 5000;
	for(int i = 0; i < k ; i++)
	{
		if(sizes[i] < curSmallest)
		{
			smallestQueue = i;
			curSmallest = sizes[i];
		}
	} 
	
	return smallestQueue;

}

__device__ __host__ int heuristic(int dim, int current, int end){//heurstics (Manhattan distance)
	
	int curX = current % dim;
	int curY = current / dim;
	int endX = end % dim;
	int endY = end / dim;
	
	return (int)(fabsf(curX - endX) + fabsf(curY - endY));
	
}

 __device__  void shiftqueue(int* array_sh, int dim, int k)
{
	
	for(int i = 0; i < dim; i++)
	{
		if(i == dim - 1)
			array_sh[dim * k + i] = -1;
		else
			array_sh[dim * k + i] = array_sh[dim * k + i + 1];
		
	}
}

__device__  int checkIfQueueEmpty(int* array_sh, int dim, int k)
{
	int check = 0;//0 if all queues empty, 1 otherwise
	for(int i = 0 ; i < k ; i++)
	{
		if(array_sh[dim * i] != -1){
			check = 1;
			break;
		}
	}
	return check;
}

__device__ __host__ void print_board(int *game, int dim){

	for (int y = 0; y < dim; y++){
                for (int x = 0; x < dim; x++){
			printf("%d ", game[x + dim*y]);
		}
		printf("\n");
	}
}

int main () {
	time_t t;
  	//int num_iter = 10;
	int dim = 6;
	int *grid = (int *) malloc(dim * dim * sizeof(int));
	int *results = (int *) malloc(dim * dim * sizeof(int));
        srand((unsigned) time(&t));
	double p = 0.5;

	for (int i = 0; i < dim*dim; i++){
		grid[i] = p < (double)rand()/(double)(RAND_MAX);
		
	}
	int * dev_grid;
	hipMalloc((void**)&dev_grid, dim*dim*sizeof(int));
	hipMemcpy(dev_grid, grid, dim*dim*sizeof(int), hipMemcpyHostToDevice);

	print_board(grid, dim);	
	//1s are spaces you can walk on
	int startPoint = -1;
	while (startPoint == - 1){
		int rando = rand() % (dim*dim);//dim*dim - 1 is highest number( 0 to dim*dim-1)
		if (grid[rando] == 1)
			startPoint = rando;
	}
	
	int endPoint = -1;
	while (endPoint == - 1){
		int rando = rand() % (dim*dim);
		if (grid[rando] == 1 && rando!=startPoint)
			endPoint = rando;
	}
	
	printf("%d %d\n", startPoint, endPoint);
	float gpu_time;
	gstart();
	traverse<<<1,5>>>(dev_grid, startPoint, endPoint, dim, 5, results);//last value is # of p-queues
	gend(&gpu_time);
	printf("GPU time = %f\n", gpu_time);
	hipDeviceSynchronize();
	return 0;


}












